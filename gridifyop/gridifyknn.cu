#include "hip/hip_runtime.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <stdio.h>
#include "./gridifyknn-inl.h"
#include <hip/hip_cooperative_groups.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>
//using namespace cooperative_groups;

#define ndim 3
#define data_ndim 4
#define CUDA_CHECK_ERROR() __cuda_check_errors(__FILE__, __LINE__)
#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)
#define max(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// See: http://codeyarns.com/2011/03/02/how-to-do-error-checking-in-cuda/
inline void
__cuda_check_errors (const char *filename, const int line_number)
{
    hipError_t err = hipDeviceSynchronize ();
    if (err != hipSuccess)
    {
        printf ("CUDA error %i at %s:%i: %s\n",
                err, filename, line_number, hipGetErrorString (err));
        exit (-1);
    }
}

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
    if (err != hipSuccess)
    {
        printf ("CUDA error %i at %s:%i: %s\n",
                err, filename, line_number, hipGetErrorString (err));
        exit (-1);
    }
}


namespace mshadow {
    namespace cuda {


// compilation workaround
// in python when use,
// import ctypes
//_ = ctypes.CDLL('additional.so')
    static __device__ double atomicAdd(double *address, double val) {
            return 0.0;
        }

    static    __device__  half::half_t
        atomicAdd(half::half_t
        * address,
        half::half_t val
        ) {
        return (half::half_t)0.0;
    }

    static __device__ float atomicAdd(float *address, float val) {
        return ::atomicAdd(address, val);
    }

    static __device__ float atomicAdd(int *address, int val) {
        return ::atomicAdd(address, val);
    }

    static __device__ float cas(double *addr, double compare, double val) {
        unsigned long long int *address_as_ull = (unsigned long long int *) addr;
        return __longlong_as_double(atomicCAS(address_as_ull,
                                              __double_as_longlong(compare),
                                              __double_as_longlong(val)));

    }

    static __device__ float cas(float *addr, float compare, float val) {
        unsigned int *address_as_uint = (unsigned int *) addr;
        return __uint_as_float(atomicCAS(address_as_uint,
                                         __float_as_uint(compare),
                                         __float_as_uint(val)));
    }

    static __device__ half::half_t
    cas(half::half_t * addr,
    half::half_t compare, half::half_t
    val) {
    // NOT IMPLEMENTED YET!
    return 0;
}




template<typename Dtype>
__global__ void gridifyKNN_kernel_build_index(
        int* out_nebidx, Dtype* out_nebidxmsk, Dtype* out_cent,
        Dtype* out_centmsk, int* out_actual_centnum, int* actual_centcount,
        const Dtype* in_data, const int* in_actual_numpoints,
        const int B,
        const int N,
        const int max_o,
        const int P,
        const int kernel_size,
        const int stride,
        const int loc,
        const float *d_coord_shift,
        const float *d_voxel_size,
        const float *d_grid_size,
        const int grid_size_vol,
        const int size,
        int *coor_to_voxelidx,
        int *voxelidx_to_coor,
        int *coor_to_pntidx,
        float *coor_to_locxyzw,
        int *coor_counter,
        unsigned long seconds
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // index of gpu thread
    int i_batch = index / N;  // index of batch
    if (i_batch >= B) { return; }
    int i_pt = index - N * i_batch;
    if (i_pt < in_actual_numpoints[i_batch]) {
        hiprandState state;
        int coor[ndim];
        const Dtype *p_pt = in_data + index * data_ndim;
        for (int j = 0; j < ndim; j++) {
            int c = floor((p_pt[j] + d_coord_shift[j]) / d_voxel_size[j]);
            if (c < 0 || c >= d_grid_size[j]) {
                return;
            }
            coor[j] = c;
        }
        int coor_indx = coor[2] * (d_grid_size[0] * d_grid_size[1])
                        + coor[1] * d_grid_size[0] + coor[0];
        int coor_indx_b = i_batch * grid_size_vol + coor_indx;

        int grid_pntidx = atomicAdd(coor_counter+coor_indx_b, 1);
        if (grid_pntidx < P) {
            coor_to_pntidx[coor_indx_b * P + grid_pntidx] = i_pt;
        } else {
            hiprand_init(index+seconds, 0, 0, &state);
            int insrtidx = ceilf(hiprand_uniform(&state) * (grid_pntidx+1)) - 1;
            if(insrtidx < P){
                coor_to_pntidx[coor_indx_b * P + insrtidx] = i_pt;
            }
        }
        if(loc == 1){
            int coor_b_idx = coor_indx_b * data_ndim;
            float weight = p_pt[3];
            atomicAdd(coor_to_locxyzw + coor_b_idx, p_pt[0] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 1, p_pt[1] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 2, p_pt[2] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 3, weight);
        }


        int voxel_idx = coor_to_voxelidx[coor_indx_b];
        //        printf("grid_size_vol: %d, coor_index: %d, i_batch %d, voxel_idx: %d ; \n", grid_size_vol, coor_index, i_batch, voxel_idx);
        if (voxel_idx == -1) {  // found an empty voxel
            Dtype old_voxel_num = atomicCAS(
                    &coor_to_voxelidx[coor_indx_b],
                    -1, 0
            );
            if (old_voxel_num == -1) {
                // CAS -> old val, if old val is -1
                // if we get -1, this thread is the one who obtain a new voxel
                // so only this thread should do the increase operator below
                int tmp = atomicAdd(out_actual_centnum + i_batch, 1); // increase the counter, return old counter
                 // increase the counter, return old counter
                if (tmp < max_o) {
                    voxelidx_to_coor[i_batch * max_o + tmp] = coor_indx;
                    out_centmsk[i_batch * max_o + tmp] = 1.0; // change center mask to 1 at new occupied voxel
                } else {
                    hiprand_init(index+2*seconds, 0, 0, &state);
                    int insrtidx = ceilf(hiprand_uniform(&state) * (tmp+1)) - 1;
                    if(insrtidx < max_o){
                        voxelidx_to_coor[i_batch * max_o + insrtidx] = coor_indx;
                    }
                }
            }
        }
    }
}

template<typename Dtype>
__global__ void gridifyKNN_kernel_query_neighs(
        int* out_nebidx, Dtype* out_nebidxmsk, Dtype* out_cent,
        Dtype* out_centmsk, int* out_actual_centnum,
        const Dtype* in_data, const int* in_actual_numpoints,
        const int B,
        const int N,
        const int max_o,
        const int P,
        const int kernel_size,
        const int stride,
        const int loc,
        const float *d_coord_shift,
        const float *d_voxel_size,
        const float *d_grid_size,
        const int grid_size_vol,
        const int size,
        int *coor_to_voxelidx,
        int *voxelidx_to_coor,
        int *coor_to_pntidx,
        float *coor_to_locxyzw,
        int *coor_counter,
        int *voxelidx_counter,
        unsigned long seconds
){
    int index = blockIdx.x * blockDim.x + threadIdx.x; // index of gpu thread
    int i_batch = index / max_o;  // index of batch
    if (i_batch >= B) { return; }
    int i_ogrid = index - i_batch * max_o;
    if(out_actual_centnum[i_batch] > max_o){
        out_actual_centnum[i_batch] = max_o;
    }
    if (i_ogrid < out_actual_centnum[i_batch]) {
        int coor_indx_b, in_data_eleweight, idx, idx_b_data;
        float xsum = 0.0, ysum = 0.0, zsum = 0.0, countweightsum = 0;
//        int coor_indx = coor[2] * (d_grid_size[0] * d_grid_size[1])
//                        + coor[1] * d_grid_size[0] + coor[0];
        int coor = voxelidx_to_coor[index];
        int coor2 = coor / (d_grid_size[0] * d_grid_size[1]);
        int coor1 = (coor - coor2* (d_grid_size[0] * d_grid_size[1])) / d_grid_size[0];
        int coor0 = coor - coor2* (d_grid_size[0] * d_grid_size[1]) - coor1 * d_grid_size[0];
        int d_coor, h_coor, w_coor;
        float total_weight = 0;
        int index_data = index * data_ndim;
        int index_P = index * P;
        int coor_indx_b_origin;

        float ux = (coor0 + 0.5) * d_voxel_size[0];
        float uy = (coor1 + 0.5) * d_voxel_size[1];
        float uz = (coor2 + 0.5) * d_voxel_size[2];

        float best[128];
        int besti[128];
        for (int l = 0; l < 64; l++){
            best[l] = FLT_MAX;
        }
        int need_P = P, amount_layer=0, amount;
        float x,y,z,dst;
        for (int layer = 0; layer < (kernel_size+1)/2; layer++){
            amount_layer = 0;
            for (int w = -layer; w < layer+1; w++) {
                for (int h = -layer; h < layer + 1; h++) {
                    for (int d = -layer; d < layer + 1; d++) {
                        if (max(max(abs(w),abs(h)),abs(d)) != layer) continue;

                        d_coor = d + coor2;
                        h_coor = h + coor1;
                        w_coor = w + coor0;
                        if (d_coor >= 0 && d_coor < d_grid_size[2] && h_coor >= 0 &&
                            h_coor < d_grid_size[1] && w_coor >= 0 && w_coor < d_grid_size[0]) {
                            coor_indx_b = i_batch * grid_size_vol + d_coor * (d_grid_size[0] * d_grid_size[1])
                                          + h_coor * d_grid_size[0] + w_coor;
                            if (layer == 0) coor_indx_b_origin = coor_indx_b;
                            amount = min(P, coor_counter[coor_indx_b]);
                            amount_layer += amount;
                            for (int g = 0; g < amount; g++) {
                                idx = coor_to_pntidx[coor_indx_b * P + g];
                                idx_b_data = (idx + N * i_batch) * data_ndim;
                                x = in_data[idx_b_data];
                                y = in_data[idx_b_data + 1];
                                z = in_data[idx_b_data + 2];
                                dst = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
                                for (int l = 0; l < P; l++) {
                                    if (dst < best[l]) {
                                        for (int j = P - 1; j > l; j--) {
                                            best[j] = best[j - 1];
                                            besti[j] = besti[j - 1];
                                        }
                                        best[l] = dst;
                                        besti[l] = idx;
                                        break;
                                    }
                                }
                            }
                        }
                    }
                }
            }
            need_P = need_P - amount_layer;
            if (need_P<=0) break;
        }

        for (int l = 0; l < P; l++){
            out_nebidx[index_P + l] = besti[l];
            idx_b_data = (besti[l] + N * i_batch) * data_ndim;
            in_data_eleweight = in_data[idx_b_data + 3];
            out_nebidxmsk[index_P + l] = 1.0;
            total_weight += in_data_eleweight;
        }

        out_cent[index_data + 3] = total_weight;
        if (need_P > 0){
            for (int j = P-need_P; j < P; j++){
                out_nebidx[index_P + j] = besti[0];
            }
        }
        if(loc==1){
            int coor_indx_b_data = coor_indx_b_origin * data_ndim;
            xsum = coor_to_locxyzw[coor_indx_b_data];
            ysum = coor_to_locxyzw[coor_indx_b_data +1];
            zsum = coor_to_locxyzw[coor_indx_b_data +2];
            countweightsum = coor_to_locxyzw[coor_indx_b_data + 3];
            out_cent[index_data] = xsum / countweightsum;
            out_cent[index_data + 1] = ysum / countweightsum;
            out_cent[index_data + 2] = zsum / countweightsum;
        }
    }
}
}  // namespace cuda

template<typename Dtype>
inline void GridifyKNNForward(Tensor<gpu, 3, int> &nebidx, // B * O * P
                           Tensor<gpu, 3, Dtype> &nebidxmsk, // B * O * P
                           Tensor<gpu, 3, Dtype> &cent, // B * O * 4
                           Tensor<gpu, 2, Dtype> &centmsk, // B * O
                           Tensor<gpu, 2, int> &actual_centnum, // B
                           const Tensor<gpu, 3, Dtype> &data,   // B * N * 4
                           const Tensor<gpu, 2, int> &actual_numpoints, // B
                           const mxnet::op::GridifyKNNParam &param
) {
    const int B = data.size(0);
    const int N = data.size(1);
    const int O = nebidx.size(1);
    const int P = nebidx.size(2);
//    printf("B: %d, N: %d, O: %d, P: %d; \n", B,N,O,P);
    // 3d voxel, ndim=3
    int *out_nebidx = nebidx.dptr_;
    Dtype *out_nebidxmsk = nebidxmsk.dptr_;
    Dtype *out_cent = cent.dptr_;
    Dtype *out_centmsk = centmsk.dptr_;
    int *out_actual_centnum = actual_centnum.dptr_;

    const Dtype *in_data = data.dptr_;
    const int *in_actual_numpoints = actual_numpoints.dptr_;

//    index_t max_p_grid;
//    index_t max_o_grid;
//    index_t kernel_size;
//    index_t stride;
//    index_t width;
//    index_t height;
//    index_t depth;
//    nnvm::Tuple<float> coord_shift;
//    nnvm::Tuple<float> voxel_size;
    int grid_size_vol = (int)(param.grid_size[0] * param.grid_size[1] * param.grid_size[2]);
    const int size = param.kernel_size * param.kernel_size * param.kernel_size;
    float *coord_shift = new float[3];
    float *voxel_size = new float[3];
    float *grid_size = new float[3];
//
//    for(int i = 0; i < size; i++){
//        d_coor_shift[i] = i / (param.kernel_size * param.kernel_size) - (param.kernel_size-1) / 2;
//        h_coor_shift[i] = (i % (param.kernel_size * param.kernel_size)) / param.kernel_size - (param.kernel_size-1) / 2;
//        w_coor_shift[i] =  i % param.kernel_size - (param.kernel_size-1) / 2;
//    }

    float *d_coord_shift, *d_voxel_size, *d_grid_size, *d_coor_to_locxyzw;
    int *d_coor_to_voxelidx, *d_voxelidx_to_coor, *d_coor_to_pntidx, *d_coor_counter, *d_voxelidx_counter, *actual_centcount;

    for (int i = 0; i < 3; ++i) {
        coord_shift[i] = param.coord_shift[i];
        voxel_size[i] = param.voxel_size[i];
        grid_size[i] = param.grid_size[i];
    }
    hipMalloc(&d_coord_shift, 3 * sizeof(float));
    hipMalloc(&d_voxel_size, 3 * sizeof(float));
    hipMalloc(&d_grid_size, 3 * sizeof(float));

    hipMemcpy(d_coord_shift, coord_shift, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_voxel_size, voxel_size, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_size, grid_size, 3 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_coor_to_locxyzw, B * grid_size_vol * data_ndim * sizeof(float));
    hipMalloc(&d_coor_to_pntidx, B * grid_size_vol * P * sizeof(int));
    hipMalloc(&d_coor_counter, B * grid_size_vol * sizeof(int));
    hipMalloc(&d_coor_to_voxelidx, B * grid_size_vol * sizeof(int));
    hipMalloc(&d_voxelidx_to_coor, B * O * sizeof(int));
    hipMalloc(&d_voxelidx_counter, B * O * sizeof(int));
    hipMalloc(&actual_centcount, B * sizeof(int));
    hipMemset(d_coor_to_locxyzw, 0, B * grid_size_vol * data_ndim * sizeof(float));
    hipMemset(d_coor_counter, 0, B * grid_size_vol * sizeof(int));
    hipMemset(d_voxelidx_counter, 0, B * O * sizeof(int));
    hipMemset(d_coor_to_voxelidx, -1, B * grid_size_vol * sizeof(int));
    hipMemset(actual_centcount, 0, B * sizeof(int));


    const int gridSize = (B * N + cuda::kMaxThreadsPerBlock - 1) / cuda::kMaxThreadsPerBlock;
//    dim3 dimGrid(sqrt(gridSize), gridSize / sqrt(gridSize) + 1);
    dim3 dimGrid(gridSize);
    dim3 dimBlock(cuda::kMaxThreadsPerBlock);
//    printf("dimGrid: %d ; dimBlock %d ;", gridSize, cuda::kMaxThreadsPerBlock);
    cuda::CheckLaunchParam(dimGrid, dimBlock, "GridifyKNNForward");
    hipStream_t stream = Stream<gpu>::GetStream(nebidx.stream_);
    timeval curTime;
    gettimeofday(&curTime, NULL);
    unsigned long seconds = curTime.tv_usec;
//    printf("seconds: %lu", seconds);
    cuda::gridifyKNN_kernel_build_index<Dtype><<<dimGrid, dimBlock>>>(
            out_nebidx, out_nebidxmsk, out_cent, out_centmsk, out_actual_centnum, actual_centcount, in_data, in_actual_numpoints,
            B, N, O, P, param.kernel_size, param.stride, param.loc, d_coord_shift, d_voxel_size, d_grid_size,
            grid_size_vol, size, d_coor_to_voxelidx, d_voxelidx_to_coor, d_coor_to_pntidx, d_coor_to_locxyzw, d_coor_counter,
            seconds);
    MSHADOW_CUDA_POST_KERNEL_CHECK(cuda::gridifyKNN_kernel_build_index);


    const int o_gridSize = (B * O + cuda::kMaxThreadsPerBlock - 1) / cuda::kMaxThreadsPerBlock;
    dim3 o_dimGrid(o_gridSize);
    dim3 o_dimBlock(cuda::kMaxThreadsPerBlock);
    cuda::gridifyKNN_kernel_query_neighs<Dtype><<<o_dimGrid, o_dimBlock>>>(
            out_nebidx, out_nebidxmsk, out_cent, out_centmsk, out_actual_centnum, in_data, in_actual_numpoints,
            B, N, O, P, param.kernel_size, param.stride, param.loc, d_coord_shift, d_voxel_size, d_grid_size,
            grid_size_vol, size, d_coor_to_voxelidx, d_voxelidx_to_coor, d_coor_to_pntidx, d_coor_to_locxyzw,
            d_coor_counter, d_voxelidx_counter, seconds);
    MSHADOW_CUDA_POST_KERNEL_CHECK(cuda::gridifyKNN_kernel_query_neighs);


    delete coord_shift;
    delete voxel_size;
    delete grid_size;
    hipFree(d_coord_shift);
    hipFree(d_voxel_size);
    hipFree(d_grid_size);
    hipFree(d_coor_to_voxelidx);
    hipFree(d_voxelidx_to_coor);
    hipFree(d_coor_to_pntidx);
    hipFree(d_coor_to_locxyzw);
    hipFree(d_coor_counter);
    hipFree(d_voxelidx_counter);
    hipFree(actual_centcount);
}

}  // namespace mshadow

namespace mxnet {
    namespace op {

        template<>
        Operator *CreateOp<gpu>(GridifyKNNParam param, int dtype) {
            Operator *op = NULL;
            MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
                    op = new GridifyKNNOp<gpu, DType>(param);
            });
            return op;
        }

    }  // namespace op
}  // namespace mxnet
